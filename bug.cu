
#include <hip/hip_runtime.h>
#include <iostream>

// Type in global namspace.
struct Type{
  int from_global_type;
};

namespace gpu {

  // Another *different* type that shares the same name.
  struct Type{
    int from_local_type;
  };

  // Kernel that uses the "gpu::Type".
  __global__ void Works(::gpu::Type t) {
    printf("%i", t.from_local_type);
  }

  // Kernel that uses the global ::Type.
  // This won't compile, but is valid C.
  __global__ void Fails(::Type t) {
    printf("%i", t.from_global_type);
  }
}