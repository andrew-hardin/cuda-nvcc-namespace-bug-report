
#include <hip/hip_runtime.h>
#include <iostream>

// Type in global namspace.
struct Type{
  int from_global_type;
};

namespace gpu {

  // Another *different* type that shares the same name.
  struct Type{
    int from_local_type;
  };

  // Kernel that uses the "gpu::Type".
  __global__ void Works(::gpu::Type t) {
    printf("%i", t.from_local_type);
  }

  // Kernel that uses the global ::Type.
  // This won't compile, but it's valid C++.
  // (e.g. remove the __global__ attribute and it'll compile just fine).
  __global__ void Fails(::Type t) {
    printf("%i", t.from_global_type);
  }
}
